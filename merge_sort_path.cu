
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <hiprand/hiprand.h>
using namespace std;

#define n 262144 // Size of array M (should be a power of 2)

// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	}
}
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

//Sort initialization
__global__
void sort_array(unsigned int *M){
	int index = blockIdx.x * 2;
	if(M[index] > M[index+1]){
		int swap = M[index];
		M[index] = M[index+1];
		M[index+1] = swap;
	}
}

//Merge path
__global__
void merge_array(unsigned int *m,unsigned int *M,int array_size){
  int index_array = blockIdx.x * array_size;
	int stride = blockDim.x;
	for(int i=threadIdx.x;i<array_size;i+=stride){

	  int size_A = array_size / 2;
	  int size_B = array_size / 2;
	  int offset;

	  int K[2],P[2],Q[2];
	  if( i> size_A ){
	      K[0] = P[1] = i - size_A;
	      K[1] = P[0] = size_A;
	    }
	    else{
	      K[0] = P[1] = 0;
	      K[1] = P[0] = i;
	    }
	    while(true){
	        offset = abs(K[1]-P[1])/2;
	        Q[0] = K[0] + offset;
	        Q[1] = K[1] - offset;
	    if (Q[1] >= 0 && Q[0] <= size_B && (Q[1] == size_A || Q[0] == 0 || M[index_array+Q[1]]>M[index_array+size_A+Q[0]-1])){
	      if(Q[0] == size_B || Q[1] == 0 || M[index_array+Q[1]-1] <= M[index_array+size_A+Q[0]]){
	        if(Q[1] < size_A && (Q[0] == size_B || M[index_array+Q[1]] <= M[index_array+size_A+Q[0]])){
	        m[index_array+i] = M[index_array+Q[1]];
	        }
	        else{
	          m[index_array+i] = M[index_array+size_A+Q[0]];
	        }
	        break;
	      }
	      else {
	        K[0] = Q[0] + 1;
	        K[1] = Q[1] - 1;
	      }
	    }
	    else{
	      P[0] = Q[0] - 1;
	      P[1] = Q[1] + 1;
	    }
		}
	}
	__syncthreads();
}

//Function to Sanity check if the array is sorted or not (CPU)
void sanity_check(unsigned int *M){
  bool sorted = true;
  for (int i = 1; i < n; i++){
      if(M[i]<M[i-1]){
        sorted = false;
				break;
      }
    }
  if(sorted==true){
    cout << "*** Is array sorted : True " << endl;
  }
  else{
      cout << "*** Is array sorted : False " << endl;
  }
}

int main () {


// *** Defining variables ***
  unsigned int *M_dev, *M_dev_next;	//Array and its copy to scale merge path procedure
	hiprandGenerator_t G; 							//Curand RNG
	clock_t timer; 										//Timer for CPU
  hipEvent_t start,stop;						//Timer for GPU
	float elapsedTime;								//Timer for GPU


// *** Allocating memory ***
	hipEventCreate(&start);
	hipEventCreate(&stop);
  testCUDA(hipMallocManaged(&M_dev, n*sizeof(unsigned int)));
	hiprandCreateGenerator(&G,HIPRAND_RNG_PSEUDO_DEFAULT);
	testCUDA(hipMallocManaged(&M_dev_next, n*sizeof(unsigned int)));


// *** Performing custom random array generation ***
  cout << "---- GENERATING RANDOM ARRAY WITH CURAND ----" << endl;
  hipEventRecord(start,0);
	hiprandGenerate(G, M_dev, n); //Generate n pseudo random unsigned int on device
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
  cout << "*** Length of array created : " << n << endl;
  cout << "*** Execution time : " << elapsedTime << " ms" << endl;


// *** Performing sort initialization ***
  cout << "\n---- SORTING ARRAYS OF LENGTH 2 TO INITIALIZE MERGE PATH ----" << endl;
  hipEventRecord(start,0);
	sort_array<<<n/2,1>>>(M_dev);
	hipEventRecord(stop,0);
  hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	cout << "*** Execution time : " << elapsedTime << " ms" << endl;


// *** Performing merge path***
	cout << "\n---- MERGING SORTED ARRAYS BY MERGE PATH ----" << endl;
	hipEventRecord(start,0);
  int nb_arrays = n/2;
  int array_size = 2;
  while (nb_arrays!=1){
		nb_arrays /= 2; //nb of array to merge on next merge_path
    array_size *= 2; //size of arrays on next merge_path
    merge_array<<<nb_arrays,min(array_size,1024)>>>(M_dev_next,M_dev,array_size);
		M_dev = M_dev_next;
		testCUDA(hipMallocManaged(&M_dev_next, n*sizeof(unsigned int)));
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  cout << "*** Execution time : " << elapsedTime << " ms" << endl;


//*** Sanity check ***
  unsigned int *M_host = (unsigned int*)malloc(n*sizeof(unsigned int));
  testCUDA(hipMemcpy(M_host, M_dev, n*sizeof(unsigned int), hipMemcpyDeviceToHost));

	/*
	//Code to vizualize array sorted
  for(int i=0;i<n;i++){
      cout <<  M_host[i] << "\t";
		}
	*/

	cout << "\n---- SANITY CHECK ----" << endl;
	timer = clock();
	sanity_check(M_host);
	timer = clock() - timer;
	cout << "*** Execution time : " << (float)timer/(CLOCKS_PER_SEC/1000) << " ms" << endl;


//*** free memory on device and host ***
	free(M_host);
  hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(M_dev);
	hipFree(M_dev_next);


  return EXIT_SUCCESS;
}
